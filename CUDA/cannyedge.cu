#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <stdlib.h>
#include "cpu_bitmap.h"
#include "bitmap_help.h"
#include <algorithm>
#include <Windows.h>

__global__ void kernel_grayscale(unsigned char*, unsigned char*, int, int);
__global__ void kernel_gaussian_blur(unsigned char*, unsigned char*, int, int);
__global__ void kernel_sobel_filter(unsigned char*, unsigned char*, unsigned char*, int, int);
__global__ void kernel_non_max_suppression(unsigned char*, unsigned char*, unsigned char*, int, int);
__global__ void kernel_hysteresis_thresholding(unsigned char*, unsigned char*, int, int);
__device__ int device_min(int, int);
__device__ int device_max(int, int);
__host__ void imgProc(unsigned char*, int, int, int);


__host__ void imgProc(unsigned char *map, int size, int width, int height) {
   
	/* Variables to time CUDA execution */
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord(start, 0);
	
    /* Allocate device memory for the result. */
    unsigned char* device_input_data;
    unsigned char* device_input_data2;
    unsigned char* device_output_data;
    unsigned char* device_edgedirection_output;
	unsigned char* host_output = new unsigned char[width * height];
		
    hipError_t err1 = hipMalloc((void**)&device_input_data, size);
	hipError_t err2 = hipMalloc((void**)&device_input_data2, size / 4);
    hipError_t err3 = hipMalloc((void**)&device_output_data, size / 4);
	hipError_t err4 = hipMalloc((void**)&device_edgedirection_output, size / 4);

	if (err1 != hipSuccess) {
		printf("%s", hipGetErrorString(err1));
		exit(EXIT_FAILURE);
	}
	if (err2 != hipSuccess) {
		printf("%s", hipGetErrorString(err2));
		exit(EXIT_FAILURE);
	}
	if (err3 != hipSuccess) {
		printf("%s", hipGetErrorString(err3));
		exit(EXIT_FAILURE);
	}
	if (err4 != hipSuccess) {
		printf("%s", hipGetErrorString(err4));
		exit(EXIT_FAILURE);
	}
	
    /* Copy the input data to the device. */
    hipMemcpy(device_input_data, map, size, hipMemcpyHostToDevice);
	
    /* Launch the kernel! */
    dim3 grid(64, 64, 1);
    dim3 block(width / 64 + 1, height / 64 + 1, 1);

	kernel_grayscale<<<grid, block>>>(device_input_data, device_output_data, height, width);
	hipMemcpy(device_input_data2, device_output_data, size / 4, hipMemcpyDeviceToDevice);
	
	kernel_gaussian_blur<<<grid, block>>>(device_input_data2, device_output_data, height, width);
	hipMemcpy(device_input_data2, device_output_data, size / 4, hipMemcpyDeviceToDevice);
	
	kernel_sobel_filter<<<grid, block>>>(device_input_data2, device_output_data, device_edgedirection_output, height, width);
	hipMemcpy(device_input_data2, device_output_data, size / 4, hipMemcpyDeviceToDevice);
	
	kernel_non_max_suppression<<<grid, block>>>(device_input_data2, device_output_data, device_edgedirection_output, height, width);
	hipMemcpy(device_input_data2, device_output_data, size / 4, hipMemcpyDeviceToDevice);
	
	kernel_hysteresis_thresholding<<<grid, block>>>(device_input_data2, device_output_data, height, width);
	hipMemcpy(host_output, device_output_data, size / 4, hipMemcpyDeviceToHost);
	
	int count = 0;
	for (int x = 0; x < width * height; x++) {
		map[count] = host_output[x];
		map[count + 1] = host_output[x];
		map[count + 2] = host_output[x];
		map[count + 3] = 0;
		count += 4;
	}
	
    hipFree(device_input_data);
    hipFree(device_input_data2);
	hipFree(device_output_data);
	hipFree(device_edgedirection_output);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Total execution time: %f ms\n", time);
}


__global__ void kernel_grayscale(unsigned char* device_input_data, unsigned char* device_output_data, int height, int width) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    /* Bound check */
    if (x < 0 || x > width || y > height || y < 0)
        return;
	
	int grayOffset = y * width + x;
	int rgbOffset = grayOffset * 4; 
	unsigned char r = device_input_data[rgbOffset];
	unsigned char g = device_input_data[rgbOffset + 1];
	unsigned char b = device_input_data[rgbOffset + 2];
	
	int grayscale = 0.21f * r + 0.71f * g + 0.07f * b;
 
    if (grayscale < 0)
        grayscale = 0;
    if (grayscale > 255)
        grayscale = 255;

    device_output_data[grayOffset] = grayscale;
}


__global__ void kernel_gaussian_blur(unsigned char* device_input_data2, unsigned char* device_output_data, int height, int width) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    /* Bound check */
    if (x < 0 || x > width || y > height || y < 0)
        return;
	
	
	int fmat[5][5] = {
		2, 4, 5, 4, 2,
		4, 9, 12, 9, 4,
		5, 12, 15, 12, 5,
		4, 9, 12, 9, 4,
		2, 4, 5, 4, 2
	};

	double factor = 1.0 / 159.0;
	double val = 0.0;

	/* Multiply every value of the filter with corresponding image pixel */
	for (int filterY = 0; filterY < 5; filterY++)
		for (int filterX = 0; filterX < 5; filterX++)
		{
			int imageX = (x - 5 / 2 + filterX + width) % width;
			int imageY = (y - 5 / 2 + filterY + height) % height;
			val += device_input_data2[imageY * width + imageX] * fmat[filterY][filterX];
		}

	/* Truncate to 0 or 255
	device_output_data[y * width + x] = device_min(device_max(int(factor * val + 1), 0), 255);
}


__global__ void kernel_sobel_filter(unsigned char* device_input_data2, unsigned char* device_output_data, unsigned char* device_edgedirection_output, int height, int width) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    /* Bound check */
    if (x < 1 || x > width - 1 || y > height - 1 || y < 1)
        return;
		
    /* To detect horizontal lines, G_x. */
    const int fmat_x[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    };
    /* To detect vertical lines, G_y */
    const int fmat_y[3][3]  = {
        {-1, -2, -1},
        {0,   0,  0},
        {1,   2,  1}
    };

    double G_x = 0;
	double G_y = 0;
	int G;

	for (int i = y - 3 / 2; i < y + 3 - 3 / 2; i++) {
		for (int j = x - 3 / 2; j < x + 3 - 3 / 2; j++) {
			G_x += (double)(fmat_x[i - y + 3 / 2][x - j + 3 / 2] * device_input_data2[i * width + j]);
			G_y += (double)(fmat_y[i - y + 3 / 2][x - j + 3 / 2] * device_input_data2[i * width + j]);
		}
	}

	/* Magnitude */
	G = sqrt(G_x * G_x + G_y * G_y);
    
    if (G < 0)
        G = 0;
    if (G > 255)
        G = 255;

    device_output_data[y * width + x] = G;
	
	float angle = atan2(G_y, G_x);

	// if negative, add 2*pi mod 2*pi for value
	if (angle < 0) {
		angle = fmod((angle + 2 * 3.14159), (2 * 3.14159));
	}

	if (angle <= 3.14159 / 8) {
		device_edgedirection_output[y * width + x] = 0;
	}
	else if (angle <= 3 * 3.14159 / 8) {
		device_edgedirection_output[y * width + x] = 45;
	}
	else if (angle <= 5 * 3.14159 / 8) {
		device_edgedirection_output[y * width + x] = 90;
	}
	else if (angle <= 7 * 3.14159 / 8) {
		device_edgedirection_output[y * width + x] = 135;
	}
	else if (angle <= 9 * 3.14159 / 8) {
		device_edgedirection_output[y * width + x] = 0;
	}
	else if (angle <= 11 * 3.14159 / 8) {
		device_edgedirection_output[y * width + x] = 45;
	}
	else if (angle <= 13 * 3.14159 / 8) {
		device_edgedirection_output[y * width + x] = 90;
	}
	else if (angle <= 15 * 3.14159 / 8) {
		device_edgedirection_output[y * width + x] = 135;
	}
	else {
		device_edgedirection_output[y * width + x] = 0;
	}
}

__global__ void kernel_non_max_suppression(unsigned char* device_input_data2, unsigned char* device_output_data, unsigned char* device_edgedirection_output, int height, int width) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    /* Bound check */
    if (x < 1 || x > width - 1 || y > height - 1 || y < 1)
        return;
		
	int POS = y * width + x;
	int N = (y - 1) * width + x;
	int NE = (y - 1) * width + (x + 1);
	int E = y * width + (x + 1);
	int SE = (y + 1) * width + (x + 1);
	int S = (y + 1) * width + x;
	int SW = (y + 1) * width + (x - 1);
	int W = y * width + (x - 1);
	int NW = (y - 1) * width + (x - 1);
	
	int val = device_input_data2[POS];
					
	switch ((int)device_edgedirection_output[y * width + x]) {
		case 0:
			if (device_input_data2[POS] <= device_input_data2[E] || device_input_data2[POS] <= device_input_data2[W]) {
				device_output_data[POS] = 0;
			}
			else {
				device_output_data[POS] = val;
			}
			break;

		case 45:
			if (device_input_data2[POS] <= device_input_data2[NE] || device_input_data2[POS] <= device_input_data2[SW]) {
				device_output_data[POS] = 0;
			}
			else {
				device_output_data[POS] = val;
			}
			break;

		case 90:
			if (device_input_data2[POS] <= device_input_data2[N] || device_input_data2[POS] <= device_input_data2[S]) {
				device_output_data[POS] = 0;
			}
			else {
				device_output_data[POS] = val;
			}
			break;
=
		case 135:
			if (device_input_data2[POS] <= device_input_data2[NW] || device_input_data2[POS] <= device_input_data2[SE]) {
				device_output_data[POS] = 0;
			}
			else {
				device_output_data[POS] = val;
			}
			break;

		default:
			device_output_data[POS] = val;
			break;
	}
}


__global__ void kernel_hysteresis_thresholding(unsigned char* device_input_data2, unsigned char* device_output_data, int height, int width) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    /* Bound check */
    if (x < 0 || x > width || y > height || y < 0)
        return;

	float lowThresh = 35;
	float highThresh = 70;

	// These variables are offset by one to avoid seg. fault errors
    // As such, this kernel ignores the outside ring of pixels
	int pos = y * width + x;

    unsigned char magnitude = device_input_data2[pos];
    
    if (magnitude >= highThresh)
        device_output_data[pos] = 255;
    else if (magnitude <= lowThresh)
        device_output_data[pos] = 0;
    else
    {
        float med = (highThresh + lowThresh) / 2;
        
        if (magnitude >= med)
            device_output_data[pos] = 255;
        else
            device_output_data[pos] = 0;
    }
}


__device__ int device_min(int a, int b) {
	return (a < b) ? a : b;
}

__device__ int device_max(int a, int b) {
	return (a < b) ? b : a;
}


int main(void) {
   char fname[50];
   FILE* infile;
   unsigned short ftype;
   tagBMFH bitHead;
   tagBMIH bitInfoHead;
   tagRGBQ *pRgb;

   printf("Please enter the .bmp file name: ");
   scanf("%s", fname);
   strcat(fname,".bmp");
   infile = fopen(fname, "rb");

   if (infile != NULL) {
      printf("File open successful.\n");
      fread(&ftype, 1, sizeof(unsigned short), infile);
      if (ftype != 0x4d42)
      {
         printf("File not .bmp format.\n");
         return 1;
      }
      fread(&bitHead, 1, sizeof(tagBMFH), infile);
      fread(&bitInfoHead, 1, sizeof(tagBMIH), infile);      
   }
   else {
      printf("File open fail.\n");
      return 1;
   }

   if (bitInfoHead.biBitCount < 24) {
      long nPlateNum = long(pow(2, double(bitInfoHead.biBitCount)));
      pRgb = (tagRGBQ *)malloc(nPlateNum * sizeof(tagRGBQ));
      memset(pRgb, 0, nPlateNum * sizeof(tagRGBQ));
      int num = fread(pRgb, 4, nPlateNum, infile);
   }

   int width = bitInfoHead.biWidth;
   int height = bitInfoHead.biHeight;
   int l_width = 4 * ((width * bitInfoHead.biBitCount + 31) / 32);
   long nData = height * l_width;
   unsigned char *pColorData = (unsigned char *)malloc(nData);
   memset(pColorData, 0, nData);
   fread(pColorData, 1, nData, infile);

   fclose(infile);
   
   CPUBitmap dataOfBmp(width, height);
   unsigned char *map = dataOfBmp.get_ptr();

   if (bitInfoHead.biBitCount < 24) {
      int k, index = 0;
      if (bitInfoHead.biBitCount == 1) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 8;
               mixIndex = pColorData[k];
               if (j % 8 < 7) mixIndex = mixIndex << (7 - (j % 8));
               mixIndex = mixIndex >> 7;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 2) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 4;
               mixIndex = pColorData[k];
               if (j % 4 < 3) mixIndex = mixIndex << (6 - 2 * (j % 4));
               mixIndex = mixIndex >> 6;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 4) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j / 2;
               mixIndex = pColorData[k];
               if (j % 2 == 0) mixIndex = mixIndex << 4;
               mixIndex = mixIndex >> 4;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 8) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j;
               mixIndex = pColorData[k];
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
       else if (bitInfoHead.biBitCount == 16) {
         for (int i = 0; i < height; i++)
            for (int j = 0; j < width; j++) {
               unsigned char mixIndex = 0;
               k = i * l_width + j * 2;
               unsigned char shortTemp = pColorData[k + 1] << 8;
               mixIndex = pColorData[k] + shortTemp;
               map[index * 4 + 0] = pRgb[mixIndex].rgbRed;
               map[index * 4 + 1] = pRgb[mixIndex].rgbGreen;
               map[index * 4 + 2] = pRgb[mixIndex].rgbBlue;
               map[index * 4 + 3] = pRgb[mixIndex].rgbReserved;
               index++;
            }
       }
   }
   else {
      int k, index = 0;
      for (int i = 0; i < height; i++)
         for (int j = 0; j < width; j++) {
            k = i * l_width + j * 3;
            map[index * 4 + 0] = pColorData[k + 2];
            map[index * 4 + 1] = pColorData[k + 1];
            map[index * 4 + 2] = pColorData[k];
            index++;
         }
   }
   
   imgProc(map, dataOfBmp.image_size(), width, height);
   dataOfBmp.display_and_exit();
   return 0;
}